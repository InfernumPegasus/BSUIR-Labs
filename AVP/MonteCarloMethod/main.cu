
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

using namespace std;
using namespace chrono;

/*
 * Var 5
 */

void PrintGPUsInfo() {
    int devices;
    hipGetDeviceCount(&devices);

    if (devices == 0) {
        std::cout << "You have no GPUs!\n";
        return;
    }

    for (int i = 0; i < devices; i++) {
        hipDeviceProp_t hipDeviceProp_t{};
        hipGetDeviceProperties(&hipDeviceProp_t, i);

        std::cout << "Device '" << hipDeviceProp_t.name << "':\n\n";
        std::cout << "max threads per block: "
                  << hipDeviceProp_t.maxThreadsPerBlock << "\n";
        std::cout << "totalGlobalMem: "
                  << hipDeviceProp_t.totalGlobalMem / 1024 / 1024
                  << " mb\n";
        std::cout << "sharedMemoryPerBlock: "
                  << hipDeviceProp_t.sharedMemPerBlock / 1024
                  << " kb.\n";
        std::cout << "compute capabilities: " << hipDeviceProp_t.major << " (major), " <<
                  hipDeviceProp_t.minor << " (minor)\n";

        auto maxGridSize = hipDeviceProp_t.maxGridSize;
        std::cout << "maxGridSize: "
                  << maxGridSize[0] << " x "
                  << maxGridSize[1] << " x "
                  << maxGridSize[2] << "\n";
        std::cout << "compute units: "
                  << hipDeviceProp_t.multiProcessorCount << "\n\n";
    }
}

using Point = float3;

constexpr Point A{0.6, 0.7, -0.1};
constexpr Point B{0.1, -1.3, -0.27};
constexpr Point C{-2.1, 0.4, -0.5};
constexpr Point D{0.3, 0.1, 0.4};

constexpr auto X_SET = {A.x, B.x, C.x, D.x};
constexpr auto Y_SET = {A.y, B.y, C.y, D.y};
constexpr auto Z_SET = {A.z, B.z, C.z, D.z};

constexpr float MIN_X = std::min(X_SET);
constexpr float MAX_X = std::max(X_SET);

constexpr float MIN_Y = std::min(Y_SET);
constexpr float MAX_Y = std::max(Y_SET);

constexpr float MIN_Z = std::min(Z_SET);
constexpr float MAX_Z = std::max(Z_SET);

constexpr float CUBE_VOLUME = abs(
        (MAX_X - MIN_X) *
        (MAX_Y - MIN_Y) *
        (MAX_Z - MIN_Z)
);

constexpr auto ITERATIONS = 100'000'000;
constexpr auto ITERATIONS_PER_THREAD = 1000;
constexpr auto TOTAL_USED_THREADS = ITERATIONS / ITERATIONS_PER_THREAD;

constexpr int MAX_THREADS_PER_BLOCK = 1024;
constexpr int THREADS_IN_WARP = 32;
constexpr int WARPS_PER_BLOCK = MAX_THREADS_PER_BLOCK / THREADS_IN_WARP;


class Plane {
public:
    constexpr Plane(
            Point p1,
            Point p2,
            Point p3) :
            a(GetDeterminant2D({p2.y - p1.y, p2.z - p1.z},
                               {p3.y - p1.y, p3.z - p1.z})),
            b(-GetDeterminant2D({p2.x - p1.x, p2.z - p1.z},
                                {p3.x - p1.x, p3.z - p1.z})),
            c(GetDeterminant2D({p2.x - p1.x, p2.y - p1.y},
                               {p3.x - p1.x, p3.y - p1.y})),
            d(-p1.x * a - p1.y * b - p1.z * c) {
        if (d < 0) {
            a *= -1;
            b *= -1;
            c *= -1;
            d *= -1;
        }
    }

    static constexpr float GetDeterminant2D(float2 r1, float2 r2) {
        return r1.x * r2.y -
               r1.y * r2.x;
    }

    static constexpr float GetDeterminant3D(float3 r1, float3 r2, float3 r3) {
        return r1.x * GetDeterminant2D({r2.y, r2.z}, {r3.y, r3.z}) -
               r2.x * GetDeterminant2D({r1.y, r1.z}, {r3.y, r3.z}) +
               r3.x * GetDeterminant2D({r1.y, r1.z}, {r2.y, r2.z});
    }

    [[nodiscard]]
    __host__
    __device__
    constexpr bool ContainsPoint(Point point) const {
        return a * point.x +
               b * point.y +
               c * point.z +
               d >= 0;
    }

public:
    [[maybe_unused]]
    void Info() const {
        std::cout << "A: " << a << "\n"
                  << "B: " << b << "\n"
                  << "C: " << c << "\n"
                  << "D: " << d << "\n";
        std::cout << "Contains {0, 0, 0}: " << std::boolalpha
                  << ContainsPoint({0, 0, 0})
                  << std::noboolalpha << "\n\n";
    }

private:
    float a, b, c, d;
};

__device__
constexpr Plane ABC(A, B, C);
__device__
constexpr Plane ABD(A, B, D);
__device__
constexpr Plane BCD(B, C, D);
__device__
constexpr Plane CAD(C, A, D);

__host__
__device__
constexpr float RandomNumber(float initial, float min, float max) {
    return initial * (max - min) + min;
}

constexpr float AnalyticComputeVolume() {
    constexpr Point AB{B.x - A.x, B.y - A.y, B.z - A.z};
    constexpr Point AC{C.x - A.x, C.y - A.y, C.z - A.z};
    constexpr Point AD{D.x - A.x, D.y - A.y, D.z - A.z};
    return abs(Plane::GetDeterminant3D(AB, AC, AD)) / 6;
}

float CPUComputeVolume() {
    int hits{};
    for (int i = 0; i < ITERATIONS; i++) {
        const auto x = RandomNumber(((float) rand() / (float) RAND_MAX),
                                    MIN_X, MAX_X);
        const auto y = RandomNumber(((float) rand() / (float) RAND_MAX),
                                    MIN_Y, MAX_Y);
        const auto z = RandomNumber(((float) rand() / (float) RAND_MAX),
                                    MIN_Z, MAX_Z);

        const Point point{x, y, z};
        if (ABC.ContainsPoint(point) &&
            ABD.ContainsPoint(point) &&
            BCD.ContainsPoint(point) &&
            CAD.ContainsPoint(point)) {
            hits++;
        }
    }
    return CUBE_VOLUME * (float) hits / (float) ITERATIONS;
}

__managed__
unsigned long long totalHits = 0;

__device__
int ShuffleDownWarp(int value) {
    constexpr auto MASK = 0xffffffff;
    for (int offset = THREADS_IN_WARP / 2; offset > 0; offset /= 2) {
        value += __shfl_down_sync(MASK, value, offset);
    }
    return value;
}

__device__
int SumBlock(int value) {
    __shared__ int reduceBuffer[WARPS_PER_BLOCK];

    const auto warpLine = threadIdx.x % THREADS_IN_WARP;
    const auto warpId = threadIdx.x / THREADS_IN_WARP;

    value = ShuffleDownWarp(value);
    if (warpLine == 0) {
        reduceBuffer[warpId] = value;
    }
    __syncthreads();

    // Loads values back only in the first warp of the first block
    value = (threadIdx.x < blockDim.x / THREADS_IN_WARP) ? reduceBuffer[warpLine] : 0;
    if (warpId == 0) {
        value = ShuffleDownWarp(value);
    }
    return value;
}

__global__
void KernelCountHits() {
    const auto threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= TOTAL_USED_THREADS) {
        return;
    }

    hiprandState randomState;
    hiprand_init(1234, threadId, 0, &randomState);

    int hits = 0;
    for (int i = 0; i < ITERATIONS_PER_THREAD; ++i) {
        const Point point{
                RandomNumber(hiprand_uniform(&randomState), MIN_X, MAX_X),
                RandomNumber(hiprand_uniform(&randomState), MIN_Y, MAX_Y),
                RandomNumber(hiprand_uniform(&randomState), MIN_Z, MAX_Z),
        };
        if (ABC.ContainsPoint(point) &&
            ABD.ContainsPoint(point) &&
            BCD.ContainsPoint(point) &&
            CAD.ContainsPoint(point)) {
            ++hits;
        }
    }

    const int blockHits = SumBlock(hits);
    if (threadIdx.x == 0) {
        atomicAdd(&totalHits, blockHits);
    }
}

float GPUComputeVolume() {
    constexpr int THREADS_PER_BLOCK = 128;
    constexpr int BLOCKS_PER_GRID = (TOTAL_USED_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    KernelCountHits<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
    return CUBE_VOLUME * (float) totalHits / (float) ITERATIONS;
}

bool SameResults(float analyticVolume, float CPUVolume, float GPUVolume) {
    return abs(GPUVolume - analyticVolume) < 0.1 &&
           abs(GPUVolume - CPUVolume) < 0.001;
}

int main() {
    srand(time(nullptr));
    PrintGPUsInfo();

//    ABC.Info();
//    ABD.Info();
//    BCD.Info();
//    CAD.Info();

    auto analyticVolume = AnalyticComputeVolume();
    std::cout << "AnalyticComputeVolume: " << analyticVolume << "\n";


    auto start = high_resolution_clock::now();
    auto cpuVolume = CPUComputeVolume();
    std::cout << "CPUComputeVolume: " << cpuVolume << "\n";
    std::cout << "CPU time: "
              << duration_cast<milliseconds>(high_resolution_clock::now() - start).count()
              << " ms\n";


    hipEvent_t eventStart, eventStop;
    hipEventCreate(&eventStart);
    hipEventCreate(&eventStop);
    hipEventRecord(eventStart);
    auto gpuVolume = GPUComputeVolume();
    std::cout << "GPUComputeVolume: " << gpuVolume << "\n";
    hipEventSynchronize(eventStop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, eventStart, eventStop);
    std::cout << "GPU time: " << milliseconds << " ms\n";

    std::cout << "Same volumes: " << std::boolalpha
              << SameResults(analyticVolume, cpuVolume, gpuVolume) << " \n";
    hipEventDestroy(eventStart);
    hipEventDestroy(eventStop);

    return 0;
}
