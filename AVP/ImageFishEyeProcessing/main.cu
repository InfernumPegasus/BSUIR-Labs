#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"
#include <cmath>
#include <stdexcept>

constexpr size_t MARKER = 0xFF00FF;
constexpr dim3 BLOCK_DIM{16, 16};

#define CUDA_ASSERT(hipError_t)                                  \
  if (hipError_t != hipSuccess) {                               \
    throw std::runtime_error{hipGetErrorString(hipError_t)};    \
  }

struct Pixel {
    uint8_t r, g, b, a;

    __host__
    __device__
    Pixel() = default;

    __device__
    Pixel(const uint8_t *image,
          const int x,
          const int y,
          const size_t width,
          const int channels) {
        r = image[x * width + y * channels];
        g = image[x * width + y * channels + 1];
        b = image[x * width + y * channels + 2];
        a = image[x * width + y * channels + 3];
    }

    __device__
    Pixel(uint8_t r,
          uint8_t g,
          uint8_t b,
          uint8_t a) : r(r), g(g), b(b), a(a) {}
};

__device__
int2 operator-(int2 a, int2 b) {
    return {a.x - b.x, a.y - b.y};
}

__device__
float2 operator-(float2 a, float2 b) {
    return {a.x - b.x, a.y - b.y};
}

__device__
float operator/(float2 a, float2 b) {
    return a.x / b.x +
           a.y / b.y;
}

__device__
Pixel operator*(Pixel a, float b) {
    a.r = static_cast<uint8_t>(static_cast<float>(a.r) * b);
    a.g = static_cast<uint8_t>(static_cast<float>(a.g) * b);
    a.b = static_cast<uint8_t>(static_cast<float>(a.b) * b);
    a.a = static_cast<uint8_t>(static_cast<float>(a.a) * b);
    return a;
}

__device__
Pixel operator+(Pixel a, Pixel b) {
    a.r = a.r + b.r;
    a.g = a.g + b.g;
    a.b = a.b + b.b;
    a.a = a.a + b.a;
    return a;
}

__device__
bool pixel_check(const uint8_t *image,
                 size_t posX) {
    return (image[posX] << 16) +
           (image[posX + 1] << 8) +
           image[posX + 2] == MARKER;
}

__host__
__device__
float normalize_value(const float value,
                      const float max) {
    return (value - ((max + 1.0f) / 2.0f)) * 2.0f / (max);
}

float calc_coefficient(const float width,
                       const float height,
                       const float radius) {
    const float normalized_source_circle_pointX = normalize_value(height / 2.0f, height);
    const float normalized_source_circle_pointY = normalize_value(width / 2.0f + radius, width);
    const float normalized_source_circle_radius = sqrt(
            normalized_source_circle_pointX * normalized_source_circle_pointX +
            normalized_source_circle_pointY * normalized_source_circle_pointY);

    const float normalized_target_circle_pointY =
            normalize_value(width / 2.0f, width);
    const float normalized_target_circle_pointX =
            normalize_value(height / 2.0f + std::min(width, height) * 0.1f, height);
    const float normalized_target_circle_radius = sqrt(
            normalized_target_circle_pointX * normalized_target_circle_pointX +
            normalized_target_circle_pointY * normalized_target_circle_pointY);

    return log(normalized_target_circle_radius) / log(normalized_source_circle_radius);
}

__device__
float aggregate_coefficient(uint4 mask) {
    if ((mask.x + mask.y + mask.z + mask.w) == 0) {
        return 0;
    } else {
        return 1.0f / static_cast<float>(mask.x + mask.y + mask.z + mask.w);
    }
}

// bilenear interpolation for float point
__device__
void interpolate(uint8_t *input,
                 const float _x,
                 const float _y,
                 const int x,
                 const int y,
                 const int width,
                 const int height,
                 const int channels,
                 const size_t pitch,
                 Pixel &pixel) {
    const int x0 = x;
    const int y0 = y;
    const int x1 = x + 1 < height ? x + 1 : x;
    const int y1 = y + 1 < width ? y + 1 : y;

    const float local_x = _x - trunc(_x);
    const float local_y = _y - trunc(_y);

    Pixel c00(input, x0, y0, pitch, channels);
    Pixel c10(input, x1, y0, pitch, channels);
    Pixel c01(input, x0, y1, pitch, channels);
    Pixel c11(input, x1, y1, pitch, channels);

    pixel = {c00 * ((1 - local_x) * (1 - local_y)) +
             c10 * (local_x * (1 - local_y)) +
             c01 * (local_y * (1 - local_x)) +
             c11 * (local_x * local_y)};
    pixel.a = 255;
}

__device__
void remove_arcs(uint8_t *image,
                 const int x,
                 const int y,
                 const int width,
                 const int height,
                 const int channels,
                 Pixel *pixel) {
    const int x0 = x - 1 > 0 ? x - 1 : x;
    const int y0 = y - 1 > 0 ? y - 1 : y;
    const int x1 = x + 1 < height ? x + 1 : x;
    const int y1 = y + 1 < width ? y + 1 : y;

    uint4 mask{1, 1, 1, 1};

    Pixel c00{image[(x0 * width + y0) * channels],
              image[(x0 * width + y0) * channels + 1],
              image[(x0 * width + y0) * channels + 2],
              image[(x0 * width + y0) * channels + 3]};

    Pixel c10{image[(x1 * width + y0) * channels],
              image[(x1 * width + y0) * channels + 1],
              image[(x1 * width + y0) * channels + 2],
              image[(x1 * width + y0) * channels + 3]};

    Pixel c01{image[(x0 * width + y1) * channels],
              image[(x0 * width + y1) * channels + 1],
              image[(x0 * width + y1) * channels + 2],
              image[(x0 * width + y1) * channels + 3]};

    Pixel c11{image[(x1 * width + y1) * channels],
              image[(x1 * width + y1) * channels + 1],
              image[(x1 * width + y1) * channels + 2],
              image[(x1 * width + y1) * channels + 3]};

    if ((c00.a + c00.b + c00.g + c00.r) == 0) {
        mask.x = 0;
    }
    if ((c10.a + c10.b + c10.g + c10.r) == 0) {
        mask.y = 0;
    }
    if ((c01.a + c01.b + c01.g + c01.r) == 0) {
        mask.z = 0;
    }
    if ((c11.a + c11.b + c11.g + c11.r) == 0) {
        mask.w = 0;
    }

    const float coefficient = aggregate_coefficient(mask);

    *pixel = {c00 * static_cast<float>(mask.x) * coefficient +
              c10 * static_cast<float>(mask.y) * coefficient +
              c01 * static_cast<float>(mask.z) * coefficient +
              c11 * static_cast<float>(mask.w) * coefficient};
    pixel->a = 255;
}

__global__
void recover(uint8_t *image,
             int width,
             int height,
             int channels,
             const bool *mask) {
    const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (y > width && x > height) {
        return;
    }
    if (!mask[x * width + y]) {
        Pixel p{};
        remove_arcs(image,
                    static_cast<int>(x), static_cast<int>(y),
                    width, height,
                    channels,
                    &p);

        const auto pos = (x * width + y) * channels;
        image[pos] = p.r;
        image[pos + 1] = p.g;
        image[pos + 2] = p.b;
        image[pos + 3] = p.a;
    }
}

unsigned int collect_accumulator(const unsigned int *accum,
                                 const int min_side) {
    unsigned int max_R = accum[0];
    for (int i = 1; i < min_side; i++) {
        if (max_R < accum[i]) {
            max_R = i;
        }
    }
    return max_R;
}

__global__
void hough_transform(unsigned char *src,
                     size_t width,
                     size_t height,
                     size_t channels,
                     unsigned int *accumulator) {
    const auto x = (threadIdx.x + blockDim.x * blockIdx.x);
    const auto y = threadIdx.y + blockDim.y * blockIdx.y;

    const auto x0 = height / 2;
    const auto y0 = width / 2;
    unsigned int R;

    if (!pixel_check(src, (x * width + y) * channels)) {
        return;
    }
    R = static_cast<int>(ceil(sqrt(static_cast<float>(((x - x0) * (x - x0)) + ((y - y0) * (y - y0))))));
    atomicAdd(accumulator + R, 1U);
}

__global__
void fish_eye(uint8_t *image,
              uint8_t *output,
              const int width,
              const int height,
              int channels,
              const float coefficient,
              const size_t im_pitch,
              bool *mask) {
    const auto x = threadIdx.x + blockDim.x * blockIdx.x;
    const auto y = threadIdx.y + blockDim.y * blockIdx.y;
    if (y > width && x > height) {
        return;
    }

    constexpr float e = 0.001f;
    const float x0 = normalize_value(static_cast<float>(x), static_cast<float>(height));
    const float y0 = normalize_value(static_cast<float>(y), static_cast<float>(width));
    const float r = sqrt(x0 * x0 + y0 * y0);
    const float theta = atan2(y0, x0);
    const float scale = min(1.0f / abs(cos(theta) + e), 1.0f / abs(sin(theta) + e));
    const float new_r = min(scale, 1.0f) * pow(r, coefficient);

    const float new_x = static_cast<float>(height) / 2.0f * new_r * cos(theta) +
                  (static_cast<float>(height) + 1.0f) / 2.0f;
    const float new_y = static_cast<float>(width) / 2.0f * new_r * sin(theta) +
                  (static_cast<float>(width) + 1.0f) / 2.0f;

    if (static_cast<unsigned int>(new_x) < height && new_x >= 0 && static_cast<unsigned int>(new_y) < width &&
        new_y >= 0) {
        Pixel p{};
        interpolate(image,
                    new_x, new_y,
                    static_cast<int>(x), static_cast<int>(y),
                    width, height,
                    channels,
                    im_pitch,
                    p);
        const int xRounded = static_cast<int>(round(new_x));
        const int yRounded = static_cast<int>(round(new_y));
        const int pos = (xRounded * width + yRounded) * channels;

        output[pos] = p.r;
        output[pos + 1] = p.g;
        output[pos + 2] = p.b;
        output[pos + 3] = p.a;
        mask[xRounded * width + yRounded] = true;
    }
}

int main() {
    int width, height, numComp;
    unsigned char *image = stbi_load("../files/zebra3.png", &width, &height, &numComp, 0);
    unsigned char *input_dev_image, *output_dev_image;

    size_t in_pitch, out_pitch;
    CUDA_ASSERT(hipMallocPitch(&input_dev_image, &in_pitch, width * numComp * sizeof(unsigned char), height))
    CUDA_ASSERT(hipMallocPitch(&output_dev_image, &out_pitch, width * numComp * sizeof(unsigned char), height))
    CUDA_ASSERT(hipMemcpy2D(input_dev_image, in_pitch, image, width * numComp * sizeof(unsigned char),
                             width * numComp * sizeof(unsigned char), height, hipMemcpyHostToDevice))

    const dim3 GRID_DIM{(height + BLOCK_DIM.x - 1) / BLOCK_DIM.x,
                        (width + BLOCK_DIM.y - 1) / BLOCK_DIM.y};

    const auto min = std::min(width, height);
    auto *accum = new unsigned int[min];
    memset(accum, 0, sizeof(unsigned int) * min);
    unsigned int *dev_accum;
    CUDA_ASSERT(hipMalloc(&dev_accum, min * sizeof(unsigned int)))
    CUDA_ASSERT(hipMemcpy(dev_accum, accum, min * sizeof(unsigned int), hipMemcpyHostToDevice))

    hough_transform<<<GRID_DIM, BLOCK_DIM>>>(input_dev_image, width, height, numComp, dev_accum);
    CUDA_ASSERT(hipMemcpy(accum, dev_accum, min * sizeof(unsigned int), hipMemcpyDeviceToHost))
    const auto radius = collect_accumulator(accum, min);
    const float coefficient = calc_coefficient(static_cast<float>(width),
                                               static_cast<float>(height),
                                               static_cast<float>(radius));

    bool *host_mask = new bool[width * height];
    bool *dev_mask;
    memset(host_mask, false, sizeof(bool) * width * height);
    CUDA_ASSERT(hipMalloc(&dev_mask, width * height * sizeof(bool)))
    CUDA_ASSERT(hipMemcpy(dev_mask, host_mask, width * height * sizeof(bool), hipMemcpyHostToDevice))
    fish_eye<<<GRID_DIM, BLOCK_DIM, sizeof(Pixel) * BLOCK_DIM.x * BLOCK_DIM.y>>>(
            input_dev_image,
            output_dev_image,
            width, height,
            numComp,
            coefficient,
            in_pitch,
            dev_mask);

    recover<<<GRID_DIM, BLOCK_DIM, sizeof(Pixel) * (BLOCK_DIM.x + 2) * (BLOCK_DIM.y + 2)>>>(
            output_dev_image,
            width, height, numComp, dev_mask);
    CUDA_ASSERT(hipMemcpy(host_mask, dev_mask, width * height * sizeof(bool), hipMemcpyDeviceToHost))

    auto *host_image = new unsigned char[width * numComp * sizeof(unsigned char) * height];
    CUDA_ASSERT(hipMemcpy2D(host_image, width * sizeof(unsigned char) * numComp, output_dev_image,
                             width * sizeof(unsigned char) * numComp, width * sizeof(unsigned char) * numComp, height,
                             hipMemcpyDeviceToHost))
    stbi_write_png("../files/result.png", width, height, 4, host_image, width * 4);
    stbi_image_free(image);

    delete[] accum;
    delete[] host_mask;

    CUDA_ASSERT(hipFree(dev_mask))
    CUDA_ASSERT(hipFree(dev_accum))
    CUDA_ASSERT(hipFree(input_dev_image))
    CUDA_ASSERT(hipFree(output_dev_image))

    return 0;
}