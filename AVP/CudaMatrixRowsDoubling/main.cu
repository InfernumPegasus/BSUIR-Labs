
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <iomanip>
#include <hiprand/hiprand.h>

using namespace std;
using namespace std::chrono;

using DataType = float;

constexpr size_t M = 7485;
constexpr size_t N = 9889;

DataType h_inputMatrix[N][M];
DataType h_outputMatrix[N][2 * M];

DataType cpu_outputMatrix[N][2 * M];

void AssertCudaSuccess(hipError_t status, std::string_view functionName) {
    if (status != hipError_t::hipSuccess) {
        throw std::runtime_error(std::string{functionName} + " error");
    }
}

void AssertCurandSuccess(hiprandStatus status, std::string_view functionName) {
    if (status != hiprandStatus::HIPRAND_STATUS_SUCCESS) {
        throw std::runtime_error(std::string{functionName} + " error");
    }
}

void CudaFillRandom(DataType *matrix) {
    hiprandGenerator_t generator;

    AssertCurandSuccess(hiprandCreateGenerator(
                                &generator, HIPRAND_RNG_PSEUDO_DEFAULT),
                        "hiprandCreateGenerator");
    AssertCurandSuccess(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL),
                        "hiprandSetPseudoRandomGeneratorSeed");
    AssertCurandSuccess(hiprandGenerateUniform(generator, matrix, N * M),
                        "hiprandGenerateUniform");
}

void PrintDeviceProperties() {
    int device;
    AssertCudaSuccess(hipGetDevice(&device),
                      "hipGetDevice");

    hipDeviceProp_t props{};
    AssertCudaSuccess(hipGetDeviceProperties(&props, device),
                      "hipGetDeviceProperties");

    std::cout << "totalGlobalMem : " << props.totalGlobalMem / 1024 / 1024 << " mb\n";
}

__global__ void KernelDoubleRows(const DataType *inputMatrix,
                                 DataType *outputMatrix,
                                 int rows,
                                 int columns) {
    auto row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        for (int col = 0; col < columns; col++) {
            auto baseRow = row * columns;

            outputMatrix[2 * (baseRow + col)] = inputMatrix[baseRow + col];
            outputMatrix[2 * (baseRow + col) + 1] = inputMatrix[baseRow + col];
        }
    }
}


void CpuDoubleRows(const DataType input_matrix[N][M],
                   DataType output_matrix[N][2 * M]) {
    auto start = high_resolution_clock::now();
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < M; col++) {
            output_matrix[row][2 * col] = input_matrix[row][col];
            output_matrix[row][2 * col + 1] = input_matrix[row][col];
        }
    }
    std::cout << "CPU time: " << std::setprecision(6) <<
              duration_cast<milliseconds>(high_resolution_clock::now() - start).count()
              << " ms\n";
}

void GpuDoubleRows(DataType *inputMatrix, DataType *outputMatrix) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 threadsPerBlock(512);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    KernelDoubleRows<<<numBlocks, threadsPerBlock>>>(
            inputMatrix, outputMatrix, N, M);

    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "GPU time: " << std::setprecision(6) << milliseconds << " ms\n";
}

bool CompareResults(const DataType cpuMatrix[N][2 * M],
                    const DataType gpuMatrix[N][2 * M]) {
    for (int i = 0; i < N; i++) {
        if (!std::equal(&cpuMatrix[i][0],
                        &cpuMatrix[i][2 * M - 1],
                        &gpuMatrix[i][0])) {
            return false;
        }
    }
    return true;
}


int main() {
    PrintDeviceProperties();

    DataType *d_inputMatrix, *d_outputMatrix;

    AssertCudaSuccess(hipMalloc((void **) &d_inputMatrix,
                                 N * M * sizeof(DataType)),
                      "hipMalloc [dev input]");

    CudaFillRandom(d_inputMatrix);

    AssertCudaSuccess(hipMemcpy(h_inputMatrix, d_inputMatrix,
                                 N * M * sizeof(int), hipMemcpyDeviceToHost),
                      "hipMemcpy [dev input]");

    CpuDoubleRows(h_inputMatrix, cpu_outputMatrix);

    AssertCudaSuccess(hipMalloc((void **) &d_outputMatrix,
                                 N * 2 * M * sizeof(DataType)),
                      "hipMalloc [dev output]");

    AssertCudaSuccess(hipMemcpy(d_inputMatrix, h_inputMatrix,
                                 N * M * sizeof(int), hipMemcpyHostToDevice),
                      "hipMemcpy [dev input]");

    GpuDoubleRows(d_inputMatrix, d_outputMatrix);

    AssertCudaSuccess(hipMemcpy(h_outputMatrix,
                                 d_outputMatrix,
                                 N * 2 * M * sizeof(int),
                                 hipMemcpyDeviceToHost),
                      "hipMemcpy [host output]");

    std::cout << "\nResult matrices equal: " << std::boolalpha <<
              CompareResults(cpu_outputMatrix,
                             h_outputMatrix) << "\n";

    AssertCudaSuccess(hipFree(d_inputMatrix),
                      "hipFree [dev input]");

    AssertCudaSuccess(hipFree(d_outputMatrix),
                      "hipFree [dev output]");

    return 0;
}
